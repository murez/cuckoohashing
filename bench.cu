#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "xxhash.hcu"
#include "utils.hcu"
#include "cuckoo_serial.hcu"
#include "cuckoo_cuda_native.hcu"
#include <string>
#include "CSVWriter.h"
using namespace csv;
void test_cuckoo_serial();
void test_cuckoo_cuda();
void bench_task1_cuckoo();
void bench_task2_cuckoo();
void bench_task3_cuckoo();
void bench_task4_cuckoo();

int main()
{
    // ha? naïve
    srand(19260817);

    // test_xxhash();
    // test_cuckoo_serial();
    // test_cuckoo_cuda();
    bench_task1_cuckoo();
    bench_task2_cuckoo();
    bench_task3_cuckoo();
    bench_task4_cuckoo();
    return 0;
}

void bench_task1_cuckoo()
{
    // create csv writer
    CsvWriter serial_writer("task1_serial.csv");
    CsvWriter cuda_writer("task1_cuda.csv");

    Record header;
    header.put("func_num");
    header.put("scale");
    header.put("mean");
    header.put("stddev");

    serial_writer.setHeader(header);
    cuda_writer.setHeader(header);

    printf("task 1\n");
    uint32_t hash_table_size = 1 << 25;

    for (int num_funcs = 2; num_funcs <= 4; ++num_funcs)
    {
        for (int scale = 10; scale <= 24; ++scale)
        {
            int n = 1 << scale;
            uint32_t *input_keys = new uint32_t[n];
            generate_unique_random(input_keys, n);

            double duration_serial[5];
            double duration_cuda[5];

            for (int rep = 0; rep < 5; rep++)
            {
                CuckooSerialHashTable ht_serial(hash_table_size, 4 * clog2(n), num_funcs);
                CuckooCudaHashTable ht_cuda(hash_table_size, 4 * clog2(n), num_funcs);
                int level_serial;
                int level_cuda;
                duration_serial[rep] = time_func([&]
                                                 { level_serial = ht_serial.insert(input_keys, n); });
                duration_cuda[rep] = time_func([&]
                                               { level_cuda = ht_cuda.insert(input_keys, n); });

                printf("hash-func-num %-10d scale %-10d repeat-times %-10d serial time: %-10lf rehash %-10d cuda time: %-10lf rehash %-10d\n", num_funcs, scale, rep, duration_serial[rep], level_serial, duration_cuda[rep], level_cuda);
            }

            double mean_serial = average(duration_serial, 5);
            double stddev_serial = standardDev(duration_serial, 5);

            double mean_cuda = average(duration_cuda, 5);
            double stddev_cuda = standardDev(duration_cuda, 5);

            Record record;
            record.put(num_funcs);
            record.put(scale);
            record.put(mean_serial);
            record.put(stddev_serial);
            serial_writer.insertRecord(record);

            Record record_cuda;
            record_cuda.put(num_funcs);
            record_cuda.put(scale);
            record_cuda.put(mean_cuda);
            record_cuda.put(stddev_cuda);
            cuda_writer.insertRecord(record_cuda);

            delete[] input_keys;
        }
    }

    serial_writer.write();
    cuda_writer.write();
}

void bench_task2_cuckoo()
{
    CsvWriter serial_writer("task2_serial.csv");
    CsvWriter cuda_writer("task2_cuda.csv");

    Record header;
    header.put("func_num");
    header.put("percent");
    header.put("mean");
    header.put("stddev");

    serial_writer.setHeader(header);
    cuda_writer.setHeader(header);

    printf("task 2\n");
    uint32_t hash_table_size = 1 << 25;
    uint32_t n = 1 << 24;
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        uint32_t *insert_values = new uint32_t[n];
        uint32_t *lookup_values = new uint32_t[n];
        bool *results = new bool[n];

        for (int percent = 0; percent <= 10; ++percent)
        {
            int bound = ceil((1 - 0.1 * percent) * n);
            double duration_serial[5];
            double duration_cuda[5];
            for (int rep = 0; rep < 5; ++rep)
            {
                generate_unique_random(insert_values, n);
                for (int i = 0; i < bound; ++i)
                    lookup_values[i] = insert_values[rand() % n];
                for (int i = bound; i < n; ++i)
                    lookup_values[i] = rand() % (int)(1.5 * n) + 1;
                CuckooSerialHashTable ht_serial(hash_table_size, 4 * clog2(n), num_funcs);
                CuckooCudaHashTable ht_cuda(hash_table_size, 4 * clog2(n), num_funcs);
                ht_serial.insert(insert_values, n);
                ht_cuda.insert(insert_values, n);

                duration_serial[rep] = time_func([&]
                                                 { ht_serial.lookup(lookup_values, results, n); });
                duration_cuda[rep] = time_func([&]
                                               { ht_cuda.lookup(lookup_values, results, n); });
                printf("hash-func-num %-10d percent %-5d repeat-times %-10d serial time: %-10lf cuda time: %-10lf\n", num_funcs, percent * 10, rep, duration_serial[rep], duration_cuda[rep]);
            }

            double mean_serial = average(duration_serial, 5);
            double stddev_serial = standardDev(duration_serial, 5);

            double mean_cuda = average(duration_cuda, 5);
            double stddev_cuda = standardDev(duration_cuda, 5);

            Record record;
            record.put(num_funcs);
            record.put(percent);
            record.put(mean_serial);
            record.put(stddev_serial);
            serial_writer.insertRecord(record);

            Record record_cuda;
            record_cuda.put(num_funcs);
            record_cuda.put(percent);
            record_cuda.put(mean_cuda);
            record_cuda.put(stddev_cuda);
            cuda_writer.insertRecord(record_cuda);
        }
        delete[] insert_values;
        delete[] lookup_values;
        delete[] results;
    }

    serial_writer.write();
    cuda_writer.write();
}

void bench_task3_cuckoo()
{
    CsvWriter serial_writer("task3_serial.csv");
    CsvWriter cuda_writer("task3_cuda.csv");

    Record header;
    header.put("func_num");
    header.put("ratios");
    header.put("mean");
    header.put("stddev");

    serial_writer.setHeader(header);
    cuda_writer.setHeader(header);

    printf("task 3\n");
    uint32_t n = 1 << 24;
    uint32_t *insert_values = new uint32_t[n];
    generate_unique_random(insert_values, n);
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        float ratios[] = {1.9, 1.8, 1.7, 1.6, 1.5, 1.4, 1.3, 1.2, 1.1, 1.05, 1.02, 1.01};
        for (int ri = 0; ri < 12; ++ri)
        {
            double duration_serial[5];
            double duration_cuda[5];

            int size = ceil(ratios[ri] * n);
            for (int rep = 0; rep < 5; ++rep)
            {
                CuckooSerialHashTable ht_serial(size, 4 * clog2(n), num_funcs);
                CuckooCudaHashTable ht_cuda(size, 4 * clog2(n), num_funcs);
                int level_serial;
                int level_cuda;
                duration_serial[rep] = time_func([&]
                                                 { level_serial = ht_serial.insert(insert_values, n); });
                duration_cuda[rep] = time_func([&]
                                               { level_cuda = ht_cuda.insert(insert_values, n); });
                printf("hash-func-num %-10d ratios %-10lf repeat-times %-10d serial time: %-10lf rehash %-10d cuda time: %-10lf rehash %-10d\n", num_funcs, ratios[ri], rep, duration_serial[rep], level_serial, duration_cuda[rep], level_cuda);
            }
            double mean_serial = average(duration_serial, 5);
            double stddev_serial = standardDev(duration_serial, 5);

            double mean_cuda = average(duration_cuda, 5);
            double stddev_cuda = standardDev(duration_cuda, 5);

            Record record;
            record.put(num_funcs);
            record.put(ratios[ri]);
            record.put(mean_serial);
            record.put(stddev_serial);
            serial_writer.insertRecord(record);

            Record record_cuda;
            record_cuda.put(num_funcs);
            record_cuda.put(ratios[ri]);
            record_cuda.put(mean_cuda);
            record_cuda.put(stddev_cuda);
            cuda_writer.insertRecord(record_cuda);
        }
    }
    serial_writer.write();
    cuda_writer.write();
    delete[] insert_values;
}

void bench_task4_cuckoo()
{
    CsvWriter serial_writer("task4_serial.csv");
    CsvWriter cuda_writer("task4_cuda.csv");

    Record header;
    header.put("func_num");
    header.put("bound_mul");
    header.put("mean");
    header.put("stddev");

    serial_writer.setHeader(header);
    cuda_writer.setHeader(header);

    printf("task 4\n");
    int n = 0x1 << 24, size = ceil(1.4 * n);
    uint32_t *insert_values = new uint32_t[n];
    generate_unique_random(insert_values, n);
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        for (int bound_mul = 1; bound_mul <= 10; ++bound_mul)
        {

            double duration_serial[5];
            double duration_cuda[5];

            for (int rep = 0; rep < 5; rep++)
            {
                CuckooSerialHashTable ht_serial(size, bound_mul * clog2(n), num_funcs);
                CuckooCudaHashTable ht_cuda(size, bound_mul * clog2(n), num_funcs);
                int level_serial;
                int level_cuda;
                duration_serial[rep] = time_func([&]
                                                 { level_serial = ht_serial.insert(insert_values, n); });
                duration_cuda[rep] = time_func([&]
                                               { level_cuda = ht_cuda.insert(insert_values, n); });
                printf("hash-func-num %-10d bound-mul %-10d repeat-times %-10d serial time: %-10lf rehash %-10d cuda time: %-10lf rehash %-10d\n", num_funcs, bound_mul, rep, duration_serial[rep], level_serial, duration_cuda[rep], level_cuda);
            }
            double mean_serial = average(duration_serial, 5);
            double stddev_serial = standardDev(duration_serial, 5);

            double mean_cuda = average(duration_cuda, 5);
            double stddev_cuda = standardDev(duration_cuda, 5);

            Record record;
            record.put(num_funcs);
            record.put(bound_mul);
            record.put(mean_serial);
            record.put(stddev_serial);
            serial_writer.insertRecord(record);

            Record record_cuda;
            record_cuda.put(num_funcs);
            record_cuda.put(bound_mul);
            record_cuda.put(mean_cuda);
            record_cuda.put(stddev_cuda);
            cuda_writer.insertRecord(record_cuda);
        }
    }
    serial_writer.write();
    cuda_writer.write();
    delete[] insert_values;
}
void test_cuckoo_serial()
{
    CuckooSerialHashTable x(1000, 12, 5);
    test_hashtable("CuckooSerialHashTable", x);
}

void test_cuckoo_cuda()
{
    CuckooCudaHashTable x(1000, 12, 5);
    test_hashtable("CuckooCudaHashTable", x);
}