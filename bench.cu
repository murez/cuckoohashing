#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "xxhash.hcu"
#include "utils.hcu"
#include "cuckoo_serial.hcu"
#include "cuckoo_cuda_native.hcu"
#include <string>
void test_cuckoo_serial();
void test_cuckoo_cuda();
void test_cuckoo_cuda_native();
void bench_task1_cuckoo_serial();
void bench_task2_cuckoo_serial();
void bench_task3_cuckoo_serial();
void bench_task4_cuckoo_serial();
int main()
{
    // ha? naïve
    srand(19260817);

    CuckooCudaHashTable q(1000, 12, 3);
    uint32_t keys[12] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11};
    q.insert(keys, 12);

    test_xxhash();
    test_cuckoo_serial();
    bench_task4_cuckoo_serial();
}

void bench_task1_cuckoo_serial()
{
    uint32_t hash_table_size = 1 << 25;
    for (int num_funcs = 2; num_funcs <= 4; ++num_funcs)
    {
        for (int scale = 10; scale <= 24; ++scale)
        {
            int n = 1 << scale;
            uint32_t *input_keys = new uint32_t[n];
            generate_unique_random(input_keys, n);
            for (int rep = 0; rep < 1; rep++)
            {
                CuckooSerialHashTable ht(hash_table_size, 4 * clog2(n), num_funcs);
                int level;
                double duation = time_func([&]
                                           { level = ht.insert(input_keys, n); });

                printf("time: %lf %d\n", duation, level);
            }
            delete[] input_keys;
        }
    }
}

void bench_task2_cuckoo_serial()
{
    uint32_t hash_table_size = 1 << 25;
    uint32_t n = 1 << 24;
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        uint32_t *insert_values = new uint32_t[n];
        uint32_t *lookup_values = new uint32_t[n];
        bool *results = new bool[n];
        for (int percent = 0; percent <= 10; ++percent)
        {
            int bound = ceil((1 - 0.1 * percent) * n);
            for (int rep = 0; rep < 5; ++rep)
            {
                generate_unique_random(insert_values, n);
                for (int i = 0; i < bound; ++i)
                    lookup_values[i] = insert_values[rand() % n];
                for (int i = bound; i < n; ++i)
                    lookup_values[i] = rand() % (int)(1.5 * n) + 1;
                CuckooSerialHashTable ht(hash_table_size, 4 * clog2(n), num_funcs);
                ht.insert(insert_values, n);
                double duation = time_func([&]
                                           { ht.lookup(lookup_values, results, n); });
                printf("time: %lf %d\n", duation, percent);
            }
        }
        delete[] insert_values;
        delete[] lookup_values;
        delete[] results;
    }
}

void bench_task3_cuckoo_serial()
{
    uint32_t n = 1 << 24;
    uint32_t *insert_values = new uint32_t[n];
    generate_unique_random(insert_values, n);
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        float ratios[] = {1.9, 1.8, 1.7, 1.6, 1.5, 1.4, 1.3, 1.2, 1.1, 1.05, 1.02, 1.01};
        for (int ri = 0; ri < 12; ++ri)
        {
            int size = ceil(ratios[ri] * n);
            for (int rep = 0; rep < 5; ++rep)
            {
                CuckooSerialHashTable ht(size, 4 * clog2(n), num_funcs);
                int level;
                double duation = time_func([&]
                                           { level = ht.insert(insert_values, n); });
                printf("%lf, %d\n", duation, level);
            }
        }
    }
}

void bench_task4_cuckoo_serial()
{
    int n = 0x1 << 24, size = ceil(1.4 * n);
    uint32_t *insert_values = new uint32_t[n];
    generate_unique_random(insert_values, n);
    for (int num_funcs = 3; num_funcs <= 4; ++num_funcs)
    {
        for (int bound_mul = 1; bound_mul <= 10; ++bound_mul)
        {
            for (int rep = 0; rep < 5; rep++)
            {
                CuckooSerialHashTable ht(size, bound_mul * clog2(n), num_funcs);
                int level;
                double duation = time_func([&]
                                           { level = ht.insert(insert_values, n); });
                printf("%lf, %d\n", duation, level);
            }
        }
    }
}
void test_cuckoo_serial()
{
    CuckooSerialHashTable x(1000, 12, 3);
    test_hashtable("CuckooSerialHashTable", x);
}

// void test_cuckoo_cuda()
// {
//     CuckooCudaHashTable x(1000, 12, 3);
//     test_hashtable("CuckooSerialHashTable", x);
// }